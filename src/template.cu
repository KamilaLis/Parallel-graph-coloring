#include "hip/hip_runtime.h"
/*
 * Katarzyna Dziewulska, Kamila Lis
 * Kolorowanie grafu metoda LF
 */

// includes, system
#include <stdlib.h>
#include <stdio.h>
#include <string.h>
#include <math.h>

// includes CUDA
#include <hip/hip_runtime.h>

// includes, project
#include <hip/hip_runtime_api.h>
#include <helper_functions.h>
// includes nvGRAPH
#include "nvgraph.h"
#include <thrust/count.h>

// struktura grafu
struct graphCSR_st {
  int nvertices;
  int nedges;
  int *source_offsets;
  int *destination_indices;
};
typedef struct graphCSR_st *graphCSR_t;

void colorLF();
int count_occur(int a[], int num_elements, int value);

graphCSR_t read_graph_DIMACS_ascii(char *file);



////////////////////////////////////////////////////////////////////////////////
//!
////////////////////////////////////////////////////////////////////////////////
__global__ void
colorLFkernel(int n, int c, int* source_offsets, int* destination_indices,
			  int* colors, int* randoms, int* out_colors)
{
	const int idx = threadIdx.x+blockIdx.x*blockDim.x;
//	printf("index: %d\n",idx);
//	printf("threats in block: %d\n", blockDim.x);

	bool f=true; // true if f you have max random

	if(idx < n){
		// ignore nodes colored earlier
		if ((colors[idx] != -1)) return;

		int ir = randoms[idx];
//		printf("my random: %d\n", ir);

		// look at neighbors to check their random number
		for (int k = source_offsets[idx]; k < source_offsets[idx+1]; k++) {
		// ignore nodes colored earlier (and yourself)
		int j = destination_indices[k];
		int jc = colors[j];
		if ((jc != -1) || (idx == j)) continue;

		int jr = randoms[j];
//		printf("neighbour random:%d\n", jr);
		if (ir <= jr) f=false;
	}
	__syncthreads();

	// assign color if you have the maximum random number
	if (f) colors[idx] = c;
	out_colors[idx] = colors[idx];
	__syncthreads();
	}
//	int i;
//    for (i = 0; i<n; i++)  printf("%d\n",colors[i]); printf("\n");
}

////////////////////////////////////////////////////////////////////////////////
// Program main
////////////////////////////////////////////////////////////////////////////////
int
main(int argc, char **argv)
{
//    // use command-line specified CUDA device, otherwise use device with highest Gflops/s
//    int cuda_device = 0;
//    cuda_device = findCudaDevice(argc, (const char **)argv);
//    hipDeviceProp_t deviceProp;
//    checkCudaErrors(hipGetDevice(&cuda_device));
//    checkCudaErrors(hipGetDeviceProperties(&deviceProp, cuda_device));
//    printf("> Detected Compute SM %d.%d hardware with %d multi-processors\n",
//           deviceProp.major, deviceProp.minor, deviceProp.multiProcessorCount);
//    if (deviceProp.major < 3)
//    {
//        printf("> nvGraph requires device SM 3.0+\n");
//        printf("> Waiving.\n");
//        exit(EXIT_WAIVED);
//    }

	graphCSR_t graph = read_graph_DIMACS_ascii("/home/klis/STUDIA/8sem/GIS/projekt/Parallel-graph-coloring/data/test.col");

//
//    StopWatchInterface *timer = 0;
//    sdkCreateTimer(&timer);
//    sdkStartTimer(&timer);

    // let's color
//	colorLF();


//    sdkStopTimer(&timer);
//    printf("Processing time: %f (ms)\n", sdkGetTimerValue(&timer));
//    sdkDeleteTimer(&timer);
    return EXIT_SUCCESS;
}

////////////////////////////////////////////////////////////////////////////////
//!
////////////////////////////////////////////////////////////////////////////////
void
colorLF()
{
	const size_t  n = 5, q = 8;
	int *source_offsets_h, *destination_indices_h;
	int i, c, *colors_h, *randoms;

    // inicjalizacja zmiennych CPU (host)
    source_offsets_h = (int*) malloc((n+1)*sizeof(int));
    destination_indices_h = (int*) malloc(q*sizeof(int));
    colors_h = (int*) malloc((n)*sizeof(int));
    randoms = (int*) malloc((n)*sizeof(int));

    // allocate mem for the result on host side
    int *out_colors_h = (int*) malloc((n)*sizeof(int));


    source_offsets_h [0] = 0;
    source_offsets_h [1] = 3;
    source_offsets_h [2] = 5;
    source_offsets_h [3] = 7;
    source_offsets_h [4] = 8;
    source_offsets_h [5] = 8;

    destination_indices_h [0] = 2;
    destination_indices_h [1] = 1;
    destination_indices_h [2] = 3;
    destination_indices_h [3] = 2;
    destination_indices_h [4] = 3;
    destination_indices_h [5] = 3;
    destination_indices_h [6] = 4;
    destination_indices_h [7] = 4;

    colors_h[0] = -1;
    colors_h [1] = -1;
    colors_h [2] = -1;
    colors_h [3] = -1;
    colors_h [4] = -1;

    randoms [0] = 0;
    randoms [1] = 1;
    randoms [2] = 2;
    randoms [3] = 3;
    randoms [4] = 4;

    // inicjalizacja zmiennych GPU (device)
    int *source_offsets_d, *destination_indices_d;
    int *colors_d, *randoms_d;
    int *out_colors_d;

    checkCudaErrors(hipMalloc((void **) &source_offsets_d, (n+1)*sizeof(int)));
    checkCudaErrors(hipMalloc((void **) &destination_indices_d, q*sizeof(int)));
    checkCudaErrors(hipMalloc((void **) &colors_d, (n)*sizeof(int)));
    checkCudaErrors(hipMalloc((void **) &out_colors_d, (n)*sizeof(int)));
    checkCudaErrors(hipMalloc((void **) &randoms_d, (n)*sizeof(int)));

    // kopiowanie na GPU
    checkCudaErrors(hipMemcpy(source_offsets_d, source_offsets_h, (n+1)*sizeof(int),
                               hipMemcpyHostToDevice));
    checkCudaErrors(hipMemcpy(destination_indices_d, destination_indices_h, q*sizeof(int),
                               hipMemcpyHostToDevice));
    checkCudaErrors(hipMemcpy(colors_d, colors_h, (n)*sizeof(int),
                               hipMemcpyHostToDevice));
    checkCudaErrors(hipMemcpy(randoms_d, randoms, (n)*sizeof(int),
                               hipMemcpyHostToDevice));

    // liczba watkow i blokow
	int num_threads = 1024;
    int num_blocks = (n/num_threads)+1;

    c=0;
    // algorytm docelowy:
    while(c <= n)
    {
    	colorLFkernel<<<1,n>>>(n, c, source_offsets_d, destination_indices_d,
    						   colors_d, randoms_d, out_colors_d);

    	++c;
        // copy result from device to host
        checkCudaErrors(hipMemcpy(out_colors_h, out_colors_d, (n)*sizeof(int),
                                   hipMemcpyDeviceToHost));
        if(count_occur(out_colors_h, n, -1) == 0) break;
    }
	// zaczekaj na wyniki obliczen GPU
	hipDeviceSynchronize();

    // wyswietlenie wyniku
    printf("out_colors_h\n");
    for (i = 0; i<n; i++)  printf("%d\n",out_colors_h[i]); printf("\n");
    printf("\nDone!\n");

    // sprzatanie
    free(source_offsets_h);
    free(destination_indices_h);
    free(randoms);
    free(colors_h);
}

////////////////////////////////////////////////////////////////////////////////
//!
////////////////////////////////////////////////////////////////////////////////
int count_occur(int a[], int num_elements, int value)
{
    int i, count = 0;
    for (i = 0; i < num_elements; i++)
    {
        if (a[i] == value)
        {
            ++count;
        }
    }
    return (count);
}
